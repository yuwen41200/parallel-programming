/**
 * Concurrent Wave Equation
 * Compilation Command: nvcc cuda1.cu -o cuda1
 * This program was originally written in serial method by the teacher.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS  1000000
#define MINPOINTS 20

static void handleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR(err) (handleError(err, __FILE__, __LINE__))

void checkParam();
__global__ void initLine(float*, float*, int);
__global__ void updateAll(float*, float*, float*, int, int);
void printResult();

int totalSteps, totalPoints, allocPoints;
float *currVal;
float *devCurrVal, *devPrevVal, *devNextVal;

int main(int argc, char *argv[]) {
	sscanf(argv[1], "%d", &totalPoints);
	sscanf(argv[2], "%d", &totalSteps);
	checkParam();

	allocPoints = totalPoints + 256;

	currVal = (float*) malloc(allocPoints * sizeof(float));
	if (!currVal)
		exit(EXIT_FAILURE);

	HANDLE_ERROR(hipMalloc((void**) &devCurrVal, allocPoints * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**) &devPrevVal, allocPoints * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**) &devNextVal, allocPoints * sizeof(float)));

	dim3 threadsPerBlock(256);
	dim3 numOfBlocks(allocPoints/256);

	printf("Initializing points on the line...\n");
	initLine<<<numOfBlocks, threadsPerBlock>>>(devPrevVal, devCurrVal, totalPoints);

	printf("Updating all points for all time steps...\n");
	updateAll<<<numOfBlocks, threadsPerBlock>>>(devPrevVal, devCurrVal, devNextVal, totalPoints, totalSteps);

	printf("Printing final results...\n");
	HANDLE_ERROR(hipMemcpy(currVal, devCurrVal, allocPoints * sizeof(float), hipMemcpyDeviceToHost));
	printResult();

	printf("\nDone.\n\n");

	hipFree(devCurrVal);
	hipFree(devPrevVal);
	hipFree(devNextVal);

	free(currVal);

	return EXIT_SUCCESS;
}

void checkParam() {
	char temp[20];
	while ((totalPoints < MINPOINTS) || (totalPoints > MAXPOINTS)) {
		printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS, MAXPOINTS);
		scanf("%s", temp);
		totalPoints = atoi(temp);
		if ((totalPoints < MINPOINTS) || (totalPoints > MAXPOINTS))
			printf("Invalid. Please enter value between %d and %d.\n", MINPOINTS, MAXPOINTS);
	}
	while ((totalSteps < 1) || (totalSteps > MAXSTEPS)) {
		printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
		scanf("%s", temp);
		totalSteps = atoi(temp);
		if ((totalSteps < 1) || (totalSteps > MAXSTEPS))
			printf("Invalid. Please enter value between 1 and %d.\n", MAXSTEPS);
	}
	printf("Using points = %d, steps = %d\n", totalPoints, totalSteps);
}

__global__ void initLine(float *__devPrevVal, float *__devCurrVal, int __totalPoints) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < __totalPoints) {
		float x = (float) i / (__totalPoints - 1);
		__devPrevVal[i] = __devCurrVal[i] = __sinf(6.28318530 * x);
	}
}

__global__ void updateAll(float *__devPrevVal, float *__devCurrVal, float *__devNextVal,
                          int __totalPoints, int __totalSteps) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < __totalPoints) {
		for (int i = 0; i < __totalSteps; i++) {
			if ((i == 0) || (i == __totalPoints - 1))
				__devNextVal[i] = 0.0;
			else
				__devNextVal[i] = 1.82 * __devCurrVal[i] - __devPrevVal[i];
			__devPrevVal[i] = __devCurrVal[i];
			__devCurrVal[i] = __devNextVal[i];
		}
	}
}

void printResult() {
	for (int i = 0; i < totalPoints; i++) {
		printf("%6.4f ", currVal[i]);
		if ((i + 1) % 10 == 0)
			printf("\n");
	}
}
